#include "hip/hip_runtime.h"
#ifdef __HIP_PLATFORM_AMD__

#include <hip/hip_runtime.h>
#include <assert.h>
#include <hip/hip_complex.h>
#include <hipblas/hipblas.h>

#else

#include "hip/hip_runtime.h"
#include ""
//#include "hip/hip_runtime.h"
// for using cublas
#include <assert.h>
#include <hip/hip_complex.h>
#include <hipblas.h>

#endif

#include <algorithm>
#include <cmath>
#include <complex>
#include <cstdio>
#include <cstdlib>
#include <cstring>

#include "memory_ops.h"
#include "util.cuh"
#include "util_func.h"
#include "util_type.h"

int get_num_device() {
    int n_gpu;
#ifdef __HIP_PLATFORM_AMD__
    hipGetDeviceCount(&n_gpu);
#else
    hipGetDeviceCount(&n_gpu);
#endif
    return n_gpu;
}

#ifdef __HIP_PLATFORM_AMD__
void set_device(unsigned int device_num) { hipSetDevice(device_num); }
#else
void set_device(unsigned int device_num) { hipSetDevice(device_num); }
#endif

int get_current_device() {
    int curr_dev_num;
#ifdef __HIP_PLATFORM_AMD__
    hipGetDevice(&curr_dev_num);
#else
    hipGetDevice(&curr_dev_num);
#endif
    return curr_dev_num;
}
inline __device__ double __shfl_down_double(
#ifdef __HIP_PLATFORM_AMD__
    double var, unsigned int srcLane, int width = 64) {
    int2 a = *reinterpret_cast<int2*>(&var);
    a.x = __shfl_down(a.x, srcLane, width);
    a.y = __shfl_down(a.y, srcLane, width);
#else
    double var, unsigned int srcLane, int width = 32) {
    int2 a = *reinterpret_cast<int2*>(&var);
    a.x = __shfl_down_sync(a.x, srcLane, width);
    a.y = __shfl_down_sync(a.y, srcLane, width);
#endif
    return *reinterpret_cast<double*>(&a);
}

inline __device__ int warpReduceSum(int val) {
    for (int offset = warpSize / 2; offset > 0; offset /= 2)
#ifdef __HIP_PLATFORM_AMD__
        val += __shfl_down(val, offset);
#else
        val += __shfl_down_sync(0xffffffff, val, offset);
#endif
    // val += __shfl_down(val, offset);
    return val;
}

// __device__ int __popcll ( unsigned long long int x )
inline __device__ int popcount64(ITYPE b) { return __popcll(b); }

//__device__ int __popc ( unsigned int  x )
inline __device__ int popcount32(unsigned int b) { return __popc(b); }

__global__ void deviceReduceWarpAtomicKernel(int* in, int* out, ITYPE N) {
    int sum = int(0);
    for (ITYPE i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
         i += blockDim.x * gridDim.x) {
        sum += in[i];
    }
    sum = warpReduceSum(sum);
    if ((threadIdx.x & (warpSize - 1)) == 0) atomicAdd(out, sum);
}

__global__ void set_computational_basis_gpu(
    ITYPE comp_basis, GTYPE* state, ITYPE dim) {
    ITYPE idx = blockIdx.x * blockDim.x + threadIdx.x;
#ifdef __HIP_PLATFORM_AMD__
    if (idx < dim) {
        state[idx] = make_hipDoubleComplex(0.0, 0.0);
    }
    if (idx == comp_basis) state[comp_basis] = make_hipDoubleComplex(1.0, 0.0);
#else
    if (idx < dim) {
        state[idx] = make_hipDoubleComplex(0.0, 0.0);
    }
    if (idx == comp_basis) state[comp_basis] = make_hipDoubleComplex(1.0, 0.0);
#endif
}

__host__ void set_computational_basis_host(ITYPE comp_basis, void* state,
    ITYPE dim, void* stream, unsigned int device_number) {
    int current_device = get_current_device();
#ifdef __HIP_PLATFORM_AMD__
    if (device_number != current_device) hipSetDevice(device_number);

    hipStream_t* hip_stream = reinterpret_cast<hipStream_t*>(stream);
#else
    if (device_number != current_device) hipSetDevice(device_number);

    hipStream_t* hip_stream = reinterpret_cast<hipStream_t*>(stream);
#endif
    GTYPE* state_gpu = reinterpret_cast<GTYPE*>(state);

    unsigned int block = dim <= 1024 ? dim : 1024;
    unsigned int grid = dim / block;

#ifdef __HIP_PLATFORM_AMD__
    set_computational_basis_gpu<<<grid, block, 0, *hip_stream>>>(
        comp_basis, state_gpu, dim);

    checkCudaErrors(hipStreamSynchronize(*hip_stream), __FILE__, __LINE__);
    checkCudaErrors(hipGetLastError(), __FILE__, __LINE__);
#else
    set_computational_basis_gpu<<<grid, block, 0, *hip_stream>>>(
        comp_basis, state_gpu, dim);

    checkCudaErrors(hipStreamSynchronize(*hip_stream), __FILE__, __LINE__);
    checkCudaErrors(hipGetLastError(), __FILE__, __LINE__);
#endif

    state = reinterpret_cast<void*>(state_gpu);
}

// copy state_gpu to state_gpu_copy
void copy_quantum_state_from_device_to_device(void* state_gpu_copy,
    const void* state_gpu, ITYPE dim, void* stream,
    unsigned int device_number) {
    int current_device = get_current_device();
#ifdef __HIP_PLATFORM_AMD__
    if (device_number != current_device) hipSetDevice(device_number);

    hipStream_t* hip_stream = reinterpret_cast<hipStream_t*>(stream);
    const GTYPE* psi_gpu = reinterpret_cast<const GTYPE*>(state_gpu);
    GTYPE* psi_gpu_copy = reinterpret_cast<GTYPE*>(state_gpu_copy);
    checkCudaErrors(hipMemcpyAsync(psi_gpu_copy, psi_gpu, dim * sizeof(GTYPE),
                        hipMemcpyDeviceToDevice, *hip_stream),
        __FILE__, __LINE__);
    checkCudaErrors(hipStreamSynchronize(*hip_stream), __FILE__, __LINE__);
#else
    if (device_number != current_device) hipSetDevice(device_number);

    hipStream_t* hip_stream = reinterpret_cast<hipStream_t*>(stream);
    const GTYPE* psi_gpu = reinterpret_cast<const GTYPE*>(state_gpu);
    GTYPE* psi_gpu_copy = reinterpret_cast<GTYPE*>(state_gpu_copy);
    checkCudaErrors(hipMemcpyAsync(psi_gpu_copy, psi_gpu, dim * sizeof(GTYPE),
                        hipMemcpyDeviceToDevice, *hip_stream),
        __FILE__, __LINE__);
    checkCudaErrors(hipStreamSynchronize(*hip_stream), __FILE__, __LINE__);
#endif
    state_gpu = reinterpret_cast<const void*>(psi_gpu);
    state_gpu_copy = reinterpret_cast<void*>(psi_gpu_copy);
}

// copy cppstate to state_gpu_copy
void copy_quantum_state_from_host_to_device(void* state_gpu_copy,
    const void* state, ITYPE dim, void* stream, unsigned int device_number) {
    int current_device = get_current_device();
#ifdef __HIP_PLATFORM_AMD__
    if (device_number != current_device) hipSetDevice(device_number);

    hipStream_t* hip_stream = reinterpret_cast<hipStream_t*>(stream);
    GTYPE* psi_gpu_copy = reinterpret_cast<GTYPE*>(state_gpu_copy);
    checkCudaErrors(hipMemcpyAsync(psi_gpu_copy, state, dim * sizeof(GTYPE),
                        hipMemcpyHostToDevice, *hip_stream),
        __FILE__, __LINE__);
    checkCudaErrors(hipStreamSynchronize(*hip_stream), __FILE__, __LINE__);
#else
    if (device_number != current_device) hipSetDevice(device_number);

    hipStream_t* hip_stream = reinterpret_cast<hipStream_t*>(stream);
    GTYPE* psi_gpu_copy = reinterpret_cast<GTYPE*>(state_gpu_copy);
    checkCudaErrors(hipMemcpyAsync(psi_gpu_copy, state, dim * sizeof(GTYPE),
                        hipMemcpyHostToDevice, *hip_stream),
        __FILE__, __LINE__);
    checkCudaErrors(hipStreamSynchronize(*hip_stream), __FILE__, __LINE__);
#endif
    state_gpu_copy = reinterpret_cast<void*>(psi_gpu_copy);
}

// this function will be removed in the future version
void copy_quantum_state_from_cppstate_host(void* state_gpu_copy,
    const CPPCTYPE* cppstate, ITYPE dim, void* stream, UINT device_number) {
    copy_quantum_state_from_host_to_device(
        state_gpu_copy, cppstate, dim, stream, device_number);
}

void copy_quantum_state_from_device_to_host(void* state_cpu_copy,
    const void* state_gpu_original, ITYPE dim, void* stream,
    unsigned int device_number) {
    int current_device = get_current_device();
#ifdef __HIP_PLATFORM_AMD__
    if (device_number != current_device) hipSetDevice(device_number);

    hipStream_t* hip_stream = reinterpret_cast<hipStream_t*>(stream);
    const GTYPE* psi_gpu = reinterpret_cast<const GTYPE*>(state_gpu_original);
    checkCudaErrors(
        hipMemcpyAsync(state_cpu_copy, psi_gpu, dim * sizeof(GTYPE),
            hipMemcpyDeviceToHost, *hip_stream),
        __FILE__, __LINE__);
    checkCudaErrors(hipStreamSynchronize(*hip_stream), __FILE__, __LINE__);
#else
    if (device_number != current_device) hipSetDevice(device_number);

    hipStream_t* hip_stream = reinterpret_cast<hipStream_t*>(stream);
    const GTYPE* psi_gpu = reinterpret_cast<const GTYPE*>(state_gpu_original);
    checkCudaErrors(
        hipMemcpyAsync(state_cpu_copy, psi_gpu, dim * sizeof(GTYPE),
            hipMemcpyDeviceToHost, *hip_stream),
        __FILE__, __LINE__);
    checkCudaErrors(hipStreamSynchronize(*hip_stream), __FILE__, __LINE__);
#endif
    state_gpu_original = reinterpret_cast<const void*>(psi_gpu);
}

// copy state_gpu to psi_cpu_copy
// this function is same as copy_quantum_state_from_device_to_host
void get_quantum_state_host(void* state_gpu, void* psi_cpu_copy, ITYPE dim,
    void* stream, unsigned int device_number) {
    int current_device = get_current_device();
#ifdef __HIP_PLATFORM_AMD__
    if (device_number != current_device) hipSetDevice(device_number);
    hipStream_t* hip_stream = reinterpret_cast<hipStream_t*>(stream);
    GTYPE* psi_gpu = reinterpret_cast<GTYPE*>(state_gpu);
    psi_cpu_copy = reinterpret_cast<CPPCTYPE*>(psi_cpu_copy);
    checkCudaErrors(
        hipMemcpyAsync(psi_cpu_copy, psi_gpu, dim * sizeof(CPPCTYPE),
            hipMemcpyDeviceToHost, *hip_stream),
        __FILE__, __LINE__);
#else
    if (device_number != current_device) hipSetDevice(device_number);
    hipStream_t* hip_stream = reinterpret_cast<hipStream_t*>(stream);
    GTYPE* psi_gpu = reinterpret_cast<GTYPE*>(state_gpu);
    psi_cpu_copy = reinterpret_cast<CPPCTYPE*>(psi_cpu_copy);
    checkCudaErrors(
        hipMemcpyAsync(psi_cpu_copy, psi_gpu, dim * sizeof(CPPCTYPE),
            hipMemcpyDeviceToHost, *hip_stream),
        __FILE__, __LINE__);
#endif
    state_gpu = reinterpret_cast<void*>(psi_gpu);
}

void print_quantum_state_host(
    void* state, ITYPE dim, unsigned int device_number) {
    int current_device = get_current_device();
#ifdef __HIP_PLATFORM_AMD__
    if (device_number != current_device) hipSetDevice(device_number);
    GTYPE* state_gpu = reinterpret_cast<GTYPE*>(state);
    CPPCTYPE* state_cpu = (CPPCTYPE*)malloc(sizeof(CPPCTYPE) * dim);
    checkCudaErrors(hipDeviceSynchronize(), __FILE__, __LINE__);
    checkCudaErrors(hipMemcpy(state_cpu, state_gpu, dim * sizeof(CPPCTYPE),
                        hipMemcpyDeviceToHost),
        __FILE__, __LINE__);
#else
    if (device_number != current_device) hipSetDevice(device_number);
    GTYPE* state_gpu = reinterpret_cast<GTYPE*>(state);
    CPPCTYPE* state_cpu = (CPPCTYPE*)malloc(sizeof(CPPCTYPE) * dim);
    checkCudaErrors(hipDeviceSynchronize(), __FILE__, __LINE__);
    checkCudaErrors(hipMemcpy(state_cpu, state_gpu, dim * sizeof(CPPCTYPE),
                        hipMemcpyDeviceToHost),
        __FILE__, __LINE__);
#endif
    for (int i = 0; i < dim; ++i) {
        std::cout << i << " : " << state_cpu[i].real() << "+i"
                  << state_cpu[i].imag() << '\n';
    }
    std::cout << '\n';
    free(state_cpu);
    state = reinterpret_cast<void*>(state);
}

ITYPE
insert_zero_to_basis_index_gsim(ITYPE basis_index, unsigned int qubit_index) {
    ITYPE temp_basis = (basis_index >> qubit_index) << (qubit_index + 1);
    return temp_basis + (basis_index & ((1ULL << qubit_index) - 1));
}

void get_Pauli_masks_partial_list_gsim(const UINT* target_qubit_index_list,
    const UINT* Pauli_operator_type_list, UINT target_qubit_index_count,
    ITYPE* bit_flip_mask, ITYPE* phase_flip_mask,
    UINT* global_phase_90rot_count, UINT* pivot_qubit_index) {
    (*bit_flip_mask) = 0;
    (*phase_flip_mask) = 0;
    (*global_phase_90rot_count) = 0;
    (*pivot_qubit_index) = 0;
    for (UINT cursor = 0; cursor < target_qubit_index_count; ++cursor) {
        UINT target_qubit_index = target_qubit_index_list[cursor];
        switch (Pauli_operator_type_list[cursor]) {
            case 0:  // I
                break;
            case 1:  // X
                (*bit_flip_mask) ^= 1ULL << target_qubit_index;
                (*pivot_qubit_index) = target_qubit_index;
                break;
            case 2:  // Y
                (*bit_flip_mask) ^= 1ULL << target_qubit_index;
                (*phase_flip_mask) ^= 1ULL << target_qubit_index;
                (*global_phase_90rot_count)++;
                (*pivot_qubit_index) = target_qubit_index;
                break;
            case 3:  // Z
                (*phase_flip_mask) ^= 1ULL << target_qubit_index;
                break;
            default:
                fprintf(stderr, "Invalid Pauli operator ID called");
                assert(0);
        }
    }
}

void get_Pauli_masks_whole_list_gsim(const UINT* Pauli_operator_type_list,
    UINT target_qubit_index_count, ITYPE* bit_flip_mask, ITYPE* phase_flip_mask,
    UINT* global_phase_90rot_count, UINT* pivot_qubit_index) {
    (*bit_flip_mask) = 0;
    (*phase_flip_mask) = 0;
    (*global_phase_90rot_count) = 0;
    (*pivot_qubit_index) = 0;
    for (UINT target_qubit_index = 0;
         target_qubit_index < target_qubit_index_count; ++target_qubit_index) {
        switch (Pauli_operator_type_list[target_qubit_index]) {
            case 0:  // I
                break;
            case 1:  // X
                (*bit_flip_mask) ^= 1ULL << target_qubit_index;
                (*pivot_qubit_index) = target_qubit_index;
                break;
            case 2:  // Y
                (*bit_flip_mask) ^= 1ULL << target_qubit_index;
                (*phase_flip_mask) ^= 1ULL << target_qubit_index;
                (*global_phase_90rot_count)++;
                (*pivot_qubit_index) = target_qubit_index;
                break;
            case 3:  // Z
                (*phase_flip_mask) ^= 1ULL << target_qubit_index;
                break;
            default:
                fprintf(stderr, "Invalid Pauli operator ID called");
                assert(0);
        }
    }
}

ITYPE* create_matrix_mask_list_gsim(
    const UINT* qubit_index_list, UINT qubit_index_count) {
    const ITYPE matrix_dim = 1ULL << qubit_index_count;
    ITYPE* mask_list = (ITYPE*)calloc((size_t)matrix_dim, sizeof(ITYPE));
    ITYPE cursor = 0;

    for (cursor = 0; cursor < matrix_dim; ++cursor) {
        for (UINT bit_cursor = 0; bit_cursor < qubit_index_count;
             ++bit_cursor) {
            if ((cursor >> bit_cursor) & 1) {
                UINT bit_index = qubit_index_list[bit_cursor];
                mask_list[cursor] ^= (1ULL << bit_index);
            }
        }
    }
    return mask_list;
}

ITYPE
create_control_mask_gsim(
    const UINT* qubit_index_list, const UINT* value_list, UINT size) {
    ITYPE mask = 0;
    for (UINT cursor = 0; cursor < size; ++cursor) {
        mask ^= (1ULL << qubit_index_list[cursor]) * value_list[cursor];
    }
    return mask;
}

UINT* create_sorted_ui_list_gsim(const UINT* array, size_t size) {
    UINT* new_array = (UINT*)calloc(size, sizeof(UINT));
    memcpy(new_array, array, size * sizeof(UINT));
    std::sort(new_array, new_array + size);
    return new_array;
}

UINT* create_sorted_ui_list_value_gsim(
    const UINT* array, size_t size, UINT value) {
    UINT* new_array = (UINT*)calloc(size + 1, sizeof(UINT));
    memcpy(new_array, array, size * sizeof(UINT));
    new_array[size] = value;
    std::sort(new_array, new_array + size + 1);
    return new_array;
}

UINT* create_sorted_ui_list_list_gsim(
    const UINT* array1, size_t size1, const UINT* array2, size_t size2) {
    UINT* new_array = (UINT*)calloc(size1 + size2, sizeof(UINT));
    memcpy(new_array, array1, size1 * sizeof(UINT));
    memcpy(new_array + size1, array2, size2 * sizeof(UINT));
    std::sort(new_array, new_array + size1 + size2);
    return new_array;
}

// C=alpha*A*B+beta*C
// in this wrapper, we assume beta is always zero!
int cublas_zgemm_wrapper(ITYPE n, CPPCTYPE alpha, const CPPCTYPE* h_A,
    const CPPCTYPE* h_B, CPPCTYPE beta, CPPCTYPE* h_C) {
    ITYPE n2 = n * n;
#ifdef __HIP_PLATFORM_AMD__
    hipblasStatus_t status;
    hipblasHandle_t handle;
#else
    hipblasStatus_t status;
    hipblasHandle_t handle;
#endif
    GTYPE* d_A;  // = make_hipDoubleComplex(0.0,0.0);
    GTYPE* d_B;  // = make_hipDoubleComplex(0,0);
    GTYPE* d_C;  // = make_hipDoubleComplex(0,0);
#ifdef __HIP_PLATFORM_AMD__
    GTYPE d_alpha = make_hipDoubleComplex(alpha.real(), alpha.imag());
    GTYPE d_beta = make_hipDoubleComplex(beta.real(), beta.imag());
#else
    GTYPE d_alpha = make_hipDoubleComplex(alpha.real(), alpha.imag());
    GTYPE d_beta = make_hipDoubleComplex(beta.real(), beta.imag());
#endif
    // int dev = 0; //findCudaDevice(argc, (const char **)argv);

    /* Initialize CUBLAS */
#ifdef __HIP_PLATFORM_AMD__
    status = hipblasCreate(&handle);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! HIPBLAS initialization error\n");
        return EXIT_FAILURE;
    }
#else
    status = hipblasCreate(&handle);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! CUBLAS initialization error\n");
        return EXIT_FAILURE;
    }
#endif

    /* Allocate device memory for the matrices */
#ifdef __HIP_PLATFORM_AMD__
    if (hipMalloc(reinterpret_cast<void**>(&d_A), n2 * sizeof(d_A[0])) !=
        hipSuccess) {
        fprintf(stderr, "!!!! device memory allocation error (allocate A)\n");
        return EXIT_FAILURE;
    }

    if (hipMalloc(reinterpret_cast<void**>(&d_B), n2 * sizeof(d_B[0])) !=
        hipSuccess) {
        fprintf(stderr, "!!!! device memory allocation error (allocate B)\n");
        return EXIT_FAILURE;
    }

    if (hipMalloc(reinterpret_cast<void**>(&d_C), n2 * sizeof(d_C[0])) !=
        hipSuccess) {
        fprintf(stderr, "!!!! device memory allocation error (allocate C)\n");
        return EXIT_FAILURE;
    }
#else
    if (hipMalloc(reinterpret_cast<void**>(&d_A), n2 * sizeof(d_A[0])) !=
        hipSuccess) {
        fprintf(stderr, "!!!! device memory allocation error (allocate A)\n");
        return EXIT_FAILURE;
    }

    if (hipMalloc(reinterpret_cast<void**>(&d_B), n2 * sizeof(d_B[0])) !=
        hipSuccess) {
        fprintf(stderr, "!!!! device memory allocation error (allocate B)\n");
        return EXIT_FAILURE;
    }

    if (hipMalloc(reinterpret_cast<void**>(&d_C), n2 * sizeof(d_C[0])) !=
        hipSuccess) {
        fprintf(stderr, "!!!! device memory allocation error (allocate C)\n");
        return EXIT_FAILURE;
    }
#endif

    /* Initialize the device matrices with the host matrices */
    // status = hipblasSetVector(n2, sizeof(h_A[0]), h_A, 1, d_A, 1);
#ifdef __HIP_PLATFORM_AMD__
    status = hipblasSetMatrix(n, n, sizeof(h_A[0]), h_A, n, d_A, n);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! device access error (write A)\n");
        return EXIT_FAILURE;
    }

    status = hipblasSetMatrix(n, n, sizeof(h_B[0]), h_B, n, d_B, n);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! device access error (write B)\n");
        return EXIT_FAILURE;
    }

    status = hipblasSetMatrix(n, n, sizeof(h_C[0]), h_C, n, d_C, n);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! device access error (write C)\n");
        return EXIT_FAILURE;
    }
    /* Performs operation using cublas */
    status = hipblasZgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, n, n, n,
        (hipblasDoubleComplex*)&d_alpha, (hipblasDoubleComplex*)d_A, n,
        (hipblasDoubleComplex*)d_B, n, (hipblasDoubleComplex*)&d_beta,
        (hipblasDoubleComplex*)d_C, n);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! kernel execution error.\n");
        return EXIT_FAILURE;
    }
#else
    status = hipblasSetMatrix(n, n, sizeof(h_A[0]), h_A, n, d_A, n);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! device access error (write A)\n");
        return EXIT_FAILURE;
    }

    // status = hipblasSetVector(n2, sizeof(h_B[0]), h_B, 1, d_B, 1);
    status = hipblasSetMatrix(n, n, sizeof(h_B[0]), h_B, n, d_B, n);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! device access error (write B)\n");
        return EXIT_FAILURE;
    }

    // status = hipblasSetVector(n2, sizeof(h_C[0]), h_C, 1, d_C, 1);
    status = hipblasSetMatrix(n, n, sizeof(h_C[0]), h_C, n, d_C, n);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! device access error (write C)\n");
        return EXIT_FAILURE;
    }
    /* Performs operation using cublas */
    status = hipblasZgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, n, n, n, &d_alpha,
        d_A, n, d_B, n, &d_beta, d_C, n);

    // status=hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha,
    // d_A, N, d_B, N, &beta, d_C, N);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! kernel execution error.\n");
        return EXIT_FAILURE;
    }
#endif

    /* Allocate host memory for reading back the result from device memory */
    CPPCTYPE* tmp_h_C =
        reinterpret_cast<CPPCTYPE*>(malloc(n2 * sizeof(h_C[0])));

    if (tmp_h_C == 0) {
        fprintf(stderr, "!!!! host memory allocation error (C)\n");
        return EXIT_FAILURE;
    }

    /* Read the result back */
#ifdef __HIP_PLATFORM_AMD__
    status = hipblasGetMatrix(n, n, sizeof(GTYPE), d_C, n, tmp_h_C, n);
    memcpy(h_C, tmp_h_C, sizeof(h_C[0]) * n2);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! device access error (read C)\n");
        return EXIT_FAILURE;
    }
    if (hipFree(d_A) != hipSuccess) {
        fprintf(stderr, "!!!! memory free error (A)\n");
        return EXIT_FAILURE;
    }

    if (hipFree(d_B) != hipSuccess) {
        fprintf(stderr, "!!!! memory free error (B)\n");
        return EXIT_FAILURE;
    }

    if (hipFree(d_C) != hipSuccess) {
        fprintf(stderr, "!!!! memory free error (C)\n");
        return EXIT_FAILURE;
    }

    /* Shutdown */
    status = hipblasDestroy(handle);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! shutdown error (A)\n");
        return EXIT_FAILURE;
    }
#else
    status = hipblasGetMatrix(n, n, sizeof(GTYPE), d_C, n, tmp_h_C, n);
    memcpy(h_C, tmp_h_C, sizeof(h_C[0]) * n2);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! device access error (read C)\n");
        return EXIT_FAILURE;
    }
    if (hipFree(d_A) != hipSuccess) {
        fprintf(stderr, "!!!! memory free error (A)\n");
        return EXIT_FAILURE;
    }

    if (hipFree(d_B) != hipSuccess) {
        fprintf(stderr, "!!!! memory free error (B)\n");
        return EXIT_FAILURE;
    }

    if (hipFree(d_C) != hipSuccess) {
        fprintf(stderr, "!!!! memory free error (C)\n");
        return EXIT_FAILURE;
    }

    /* Shutdown */
    status = hipblasDestroy(handle);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! shutdown error (A)\n");
        return EXIT_FAILURE;
    }
#endif
    return 0;
}

// C=alpha*A*x+beta*y
// in this wrapper, we assume beta is always zero!
int cublas_zgemv_wrapper(ITYPE n, CPPCTYPE alpha, const CPPCTYPE* h_A,
    const CPPCTYPE* h_x, CPPCTYPE beta, CPPCTYPE* h_y) {
    ITYPE n2 = n * n;
#ifdef __HIP_PLATFORM_AMD__
    hipblasStatus_t status;
    hipblasHandle_t handle;
#else
    hipblasStatus_t status;
    hipblasHandle_t handle;
#endif
    GTYPE* d_A;
    GTYPE* d_x;
    GTYPE* d_y;
#ifdef __HIP_PLATFORM_AMD__
    GTYPE d_alpha = make_hipDoubleComplex(alpha.real(), alpha.imag());
    GTYPE d_beta = make_hipDoubleComplex(beta.real(), beta.imag());
#else
    GTYPE d_alpha = make_hipDoubleComplex(alpha.real(), alpha.imag());
    GTYPE d_beta = make_hipDoubleComplex(beta.real(), beta.imag());
#endif
    // int dev = 0; //findCudaDevice(argc, (const char **)argv);

    /* Initialize CUBLAS */
#ifdef __HIP_PLATFORM_AMD__
    printf("simpleCUBLAS test running..\n");
    status = hipblasCreate(&handle);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! CUBLAS initialization error\n");
        return EXIT_FAILURE;
    }

    /* Allocate device memory for the matrices */
    if (hipMalloc(reinterpret_cast<void**>(&d_A), n2 * sizeof(d_A[0])) !=
        hipSuccess) {
        fprintf(stderr, "!!!! device memory allocation error (allocate A)\n");
        return EXIT_FAILURE;
    }

    if (hipMalloc(reinterpret_cast<void**>(&d_x), n * sizeof(d_x[0])) !=
        hipSuccess) {
        fprintf(stderr, "!!!! device memory allocation error (allocate x)\n");
        return EXIT_FAILURE;
    }

    if (hipMalloc(reinterpret_cast<void**>(&d_y), n * sizeof(d_y[0])) !=
        hipSuccess) {
        fprintf(stderr, "!!!! device memory allocation error (allocate y)\n");
        return EXIT_FAILURE;
    }
#else
    printf("simpleCUBLAS test running..\n");
    status = hipblasCreate(&handle);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! CUBLAS initialization error\n");
        return EXIT_FAILURE;
    }

    /* Allocate device memory for the matrices */
    if (hipMalloc(reinterpret_cast<void**>(&d_A), n2 * sizeof(d_A[0])) !=
        hipSuccess) {
        fprintf(stderr, "!!!! device memory allocation error (allocate A)\n");
        return EXIT_FAILURE;
    }

    if (hipMalloc(reinterpret_cast<void**>(&d_x), n * sizeof(d_x[0])) !=
        hipSuccess) {
        fprintf(stderr, "!!!! device memory allocation error (allocate x)\n");
        return EXIT_FAILURE;
    }

    if (hipMalloc(reinterpret_cast<void**>(&d_y), n * sizeof(d_y[0])) !=
        hipSuccess) {
        fprintf(stderr, "!!!! device memory allocation error (allocate y)\n");
        return EXIT_FAILURE;
    }
#endif

    /* Initialize the device matrices with the host matrices */
    // status = hipblasSetVector(n2, sizeof(h_A[0]), h_A, 1, d_A, 1);
#ifdef __HIP_PLATFORM_AMD__
    status = hipblasSetMatrix(n, n, sizeof(h_A[0]), h_A, n, d_A, n);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! device access error (write A)\n");
        return EXIT_FAILURE;
    }

    status = hipblasSetVector(n, sizeof(h_x[0]), h_x, 1, d_x, 1);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! device access error (write x)\n");
        return EXIT_FAILURE;
    }

    status = hipblasSetVector(n, sizeof(h_y[0]), h_y, 1, d_y, 1);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! device access error (write C)\n");
        return EXIT_FAILURE;
    }
    /* Performs operation using cublas */
    status = hipblasZgemv(
        handle, HIPBLAS_OP_T, n, n, (hipblasDoubleComplex*)&d_alpha,
        (hipblasDoubleComplex*)d_A, n, (hipblasDoubleComplex*)d_x, 1,
        (hipblasDoubleComplex*)&d_beta, (hipblasDoubleComplex*)d_y, 1);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! kernel execution error.\n");
        return EXIT_FAILURE;
    }
#else
    status = hipblasSetMatrix(n, n, sizeof(h_A[0]), h_A, n, d_A, n);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! device access error (write A)\n");
        return EXIT_FAILURE;
    }

    status = hipblasSetVector(n, sizeof(h_x[0]), h_x, 1, d_x, 1);
    // status = hipblasSetMatrix(n, n, sizeof(h_B[0]), h_B, n, d_B, n);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! device access error (write x)\n");
        return EXIT_FAILURE;
    }

    status = hipblasSetVector(n, sizeof(h_y[0]), h_y, 1, d_y, 1);
    // status = hipblasSetMatrix(n, n, sizeof(h_C[0]), h_C, n, d_C, n);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! device access error (write C)\n");
        return EXIT_FAILURE;
    }
    /* Performs operation using cublas */
    status = hipblasZgemv(
        handle, HIPBLAS_OP_T, n, n, &d_alpha, d_A, n, d_x, 1, &d_beta, d_y, 1);
    /*
    hipblasStatus_t hipblasZgemv(hipblasHandle_t handle, hipblasOperation_t trans,
                               int m, int n,
                               const hipDoubleComplex *alpha,
                               const hipDoubleComplex *A, int lda,
                               const hipDoubleComplex *x, int incx,
                               const hipDoubleComplex *beta,
                               hipDoubleComplex *y, int incy)
    */
    // status=hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha,
    // d_A, N, d_B, N, &beta, d_C, N);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! kernel execution error.\n");
        return EXIT_FAILURE;
    }
#endif

    /* Allocate host memory for reading back the result from device memory */
    CPPCTYPE* tmp_h_y = reinterpret_cast<CPPCTYPE*>(malloc(n * sizeof(h_y[0])));

    if (tmp_h_y == 0) {
        fprintf(stderr, "!!!! host memory allocation error (y)\n");
        return EXIT_FAILURE;
    }

    /* Read the result back */
#ifdef __HIP_PLATFORM_AMD__
    status = hipblasGetVector(n, sizeof(GTYPE), d_y, 1, tmp_h_y, 1);
    memcpy(h_y, tmp_h_y, sizeof(h_y[0]) * n);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! device access error (read C)\n");
        return EXIT_FAILURE;
    }
    if (hipFree(d_A) != hipSuccess) {
        fprintf(stderr, "!!!! memory free error (A)\n");
        return EXIT_FAILURE;
    }

    if (hipFree(d_x) != hipSuccess) {
        fprintf(stderr, "!!!! memory free error (x)\n");
        return EXIT_FAILURE;
    }

    if (hipFree(d_y) != hipSuccess) {
        fprintf(stderr, "!!!! memory free error (y)\n");
        return EXIT_FAILURE;
    }

    /* Shutdown */
    status = hipblasDestroy(handle);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! shutdown error (A)\n");
        return EXIT_FAILURE;
    }
#else
    status = hipblasGetVector(n, sizeof(GTYPE), d_y, 1, tmp_h_y, 1);
    /*
    hipblasStatus_t hipblasGetVector(int n, int elemSize, const void *x, int incx,
    void *y, int incy)
    */
    memcpy(h_y, tmp_h_y, sizeof(h_y[0]) * n);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! device access error (read C)\n");
        return EXIT_FAILURE;
    }
    if (hipFree(d_A) != hipSuccess) {
        fprintf(stderr, "!!!! memory free error (A)\n");
        return EXIT_FAILURE;
    }

    if (hipFree(d_x) != hipSuccess) {
        fprintf(stderr, "!!!! memory free error (x)\n");
        return EXIT_FAILURE;
    }

    if (hipFree(d_y) != hipSuccess) {
        fprintf(stderr, "!!!! memory free error (y)\n");
        return EXIT_FAILURE;
    }

    /* Shutdown */
    status = hipblasDestroy(handle);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! shutdown error (A)\n");
        return EXIT_FAILURE;
    }
#endif
    return 0;
}

// we assume state has already allocated at device
int cublas_zgemv_wrapper(ITYPE n, const CPPCTYPE* h_matrix, GTYPE* d_state) {
    ITYPE n2 = n * n;
#ifdef __HIP_PLATFORM_AMD__
    hipblasStatus_t status;
    hipblasHandle_t handle;
#else
    hipblasStatus_t status;
    hipblasHandle_t handle;
#endif
    GTYPE* d_matrix;
    GTYPE* d_y;  // this will include the answer of the state.
#ifdef __HIP_PLATFORM_AMD__
    GTYPE d_alpha = make_hipDoubleComplex(1.0, 0.0);
    GTYPE d_beta = make_hipDoubleComplex(0.0, 0.0);
#else
    GTYPE d_alpha = make_hipDoubleComplex(1.0, 0.0);
    GTYPE d_beta = make_hipDoubleComplex(0.0, 0.0);
#endif
    // int dev = 0;

    /* Initialize CUBLAS */
#ifdef __HIP_PLATFORM_AMD__
    status = hipblasCreate(&handle);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! CUBLAS initialization error\n");
        return EXIT_FAILURE;
    }

    /* Allocate device memory for the matrices */
    if (hipMalloc(reinterpret_cast<void**>(&d_matrix),
            n2 * sizeof(d_matrix[0])) != hipSuccess) {
        fprintf(stderr, "!!!! device memory allocation error (allocate A)\n");
        return EXIT_FAILURE;
    }

    if (hipMalloc(reinterpret_cast<void**>(&d_y), n * sizeof(d_y[0])) !=
        hipSuccess) {
        fprintf(stderr, "!!!! device memory allocation error (allocate y)\n");
        return EXIT_FAILURE;
    }
#else
    status = hipblasCreate(&handle);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! CUBLAS initialization error\n");
        return EXIT_FAILURE;
    }

    /* Allocate device memory for the matrices */
    if (hipMalloc(reinterpret_cast<void**>(&d_matrix),
            n2 * sizeof(d_matrix[0])) != hipSuccess) {
        fprintf(stderr, "!!!! device memory allocation error (allocate A)\n");
        return EXIT_FAILURE;
    }

    if (hipMalloc(reinterpret_cast<void**>(&d_y), n * sizeof(d_y[0])) !=
        hipSuccess) {
        fprintf(stderr, "!!!! device memory allocation error (allocate y)\n");
        return EXIT_FAILURE;
    }
#endif
    // hipMemset(&d_y, 0, sizeof(d_y[0])*n);
    /* Initialize the device matrices with the host matrices */
#ifdef __HIP_PLATFORM_AMD__
    status =
        hipblasSetMatrix(n, n, sizeof(h_matrix[0]), h_matrix, n, d_matrix, n);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! device access error (write A)\n");
        return EXIT_FAILURE;
    }

    /* Performs operation using cublas */
    status = hipblasZgemv(handle, HIPBLAS_OP_T, n, n, (hipblasDoubleComplex*)&d_alpha,
        (hipblasDoubleComplex*)d_matrix, n, (hipblasDoubleComplex*)d_state, 1,
        (hipblasDoubleComplex*)&d_beta, (hipblasDoubleComplex*)d_y, 1);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! kernel execution error.\n");
        return EXIT_FAILURE;
    }

    hipMemcpy(d_state, d_y, n * sizeof(GTYPE), hipMemcpyDeviceToDevice);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! device access error (read C)\n");
        return EXIT_FAILURE;
    }
    if (hipFree(d_matrix) != hipSuccess) {
        fprintf(stderr, "!!!! memory free error (A)\n");
        return EXIT_FAILURE;
    }

    if (hipFree(d_y) != hipSuccess) {
        fprintf(stderr, "!!!! memory free error (y)\n");
        return EXIT_FAILURE;
    }

    /* Shutdown */
    status = hipblasDestroy(handle);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! shutdown error (A)\n");
        return EXIT_FAILURE;
    }
#else
    status =
        hipblasSetMatrix(n, n, sizeof(h_matrix[0]), h_matrix, n, d_matrix, n);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! device access error (write A)\n");
        return EXIT_FAILURE;
    }

    /* Performs operation using cublas */
    status = hipblasZgemv(handle, HIPBLAS_OP_T, n, n, &d_alpha, d_matrix, n,
        d_state, 1, &d_beta, d_y, 1);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! kernel execution error.\n");
        return EXIT_FAILURE;
    }

    hipMemcpy(d_state, d_y, n * sizeof(GTYPE), hipMemcpyDeviceToDevice);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! device access error (read C)\n");
        return EXIT_FAILURE;
    }
    if (hipFree(d_matrix) != hipSuccess) {
        fprintf(stderr, "!!!! memory free error (A)\n");
        return EXIT_FAILURE;
    }

    if (hipFree(d_y) != hipSuccess) {
        fprintf(stderr, "!!!! memory free error (y)\n");
        return EXIT_FAILURE;
    }

    /* Shutdown */
    status = hipblasDestroy(handle);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! shutdown error (A)\n");
        return EXIT_FAILURE;
    }
#endif
    return 0;
}

// we assume state and matrix has already allocated at device
int cublas_zgemv_wrapper(ITYPE n, const GTYPE* d_matrix, GTYPE* d_state) {
    // ITYPE n2 = n*n;
#ifdef __HIP_PLATFORM_AMD__
    hipblasStatus_t status;
    hipblasHandle_t handle;
    GTYPE* d_y;  // this will include the answer of the state.
    GTYPE d_alpha = make_hipDoubleComplex(1.0, 0.0);
    GTYPE d_beta = make_hipDoubleComplex(0.0, 0.0);
#else
    hipblasStatus_t status;
    hipblasHandle_t handle;
    GTYPE* d_y;  // this will include the answer of the state.
    GTYPE d_alpha = make_hipDoubleComplex(1.0, 0.0);
    GTYPE d_beta = make_hipDoubleComplex(0.0, 0.0);
#endif
    // int dev = 0;

    /* Initialize CUBLAS */
#ifdef __HIP_PLATFORM_AMD__
    status = hipblasCreate(&handle);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! CUBLAS initialization error\n");
        return EXIT_FAILURE;
    }

    if (hipMalloc(reinterpret_cast<void**>(&d_y), n * sizeof(d_y[0])) !=
        hipSuccess) {
        fprintf(stderr, "!!!! device memory allocation error (allocate y)\n");
        return EXIT_FAILURE;
    }

    /* Performs operation using cublas */
    status = hipblasZgemv(handle, HIPBLAS_OP_T, n, n, (hipblasDoubleComplex*)&d_alpha,
        (hipblasDoubleComplex*)d_matrix, n, (hipblasDoubleComplex*)d_state, 1,
        (hipblasDoubleComplex*)&d_beta, (hipblasDoubleComplex*)d_y, 1);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! kernel execution error.\n");
        return EXIT_FAILURE;
    }

    hipMemcpy(d_state, d_y, n * sizeof(GTYPE), hipMemcpyDeviceToDevice);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! device access error (read C)\n");
        return EXIT_FAILURE;
    }

    if (hipFree(d_y) != hipSuccess) {
        fprintf(stderr, "!!!! memory free error (y)\n");
        return EXIT_FAILURE;
    }

    /* Shutdown */
    status = hipblasDestroy(handle);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! shutdown error (A)\n");
        return EXIT_FAILURE;
    }
#else
    status = hipblasCreate(&handle);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! CUBLAS initialization error\n");
        return EXIT_FAILURE;
    }

    if (hipMalloc(reinterpret_cast<void**>(&d_y), n * sizeof(d_y[0])) !=
        hipSuccess) {
        fprintf(stderr, "!!!! device memory allocation error (allocate y)\n");
        return EXIT_FAILURE;
    }
    // hipMemset(&d_y, 0, sizeof(d_y[0])*n);

    /* Performs operation using cublas */
    status = hipblasZgemv(handle, HIPBLAS_OP_T, n, n, &d_alpha, d_matrix, n,
        d_state, 1, &d_beta, d_y, 1);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! kernel execution error.\n");
        return EXIT_FAILURE;
    }

    hipMemcpy(d_state, d_y, n * sizeof(GTYPE), hipMemcpyDeviceToDevice);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! device access error (read C)\n");
        return EXIT_FAILURE;
    }

    if (hipFree(d_y) != hipSuccess) {
        fprintf(stderr, "!!!! memory free error (y)\n");
        return EXIT_FAILURE;
    }

    /* Shutdown */
    status = hipblasDestroy(handle);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! shutdown error (A)\n");
        return EXIT_FAILURE;
    }
#endif
    return 0;
}
